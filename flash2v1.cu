#include "hip/hip_runtime.h"
//%%writefile flash2.cu
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>
#include <time.h>
#include <cmath>

namespace cg = cooperative_groups;

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define CURAND_CHECK(ans) { curandAssert((ans), __FILE__, __LINE__); }
inline void curandAssert(hiprandStatus_t code, const char *file, int line, bool abort=true) {
   if (code != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "CURANDassert: %d %s %d\n", code, file, line);
      if (abort) exit(code);
   }
}

__global__ void scale_shift_kernel(float* input, float* output, int n, float min_val, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = min_val + input[idx] * scale;
    }
}

// Forward FlashAttention-2
__global__ void flash_attention_2_forward(
    const float* __restrict__ Q,
    const float* __restrict__ K,
    const float* __restrict__ V,
    float* __restrict__ O,
    float* __restrict__ l,
    float* __restrict__ m,
    const int seq_len,
    const int embed_dim,
    const int block_size_row,
    const int block_size_col,
    const float scale)
{
    cg::thread_block block = cg::this_thread_block();
    int bx = blockIdx.x;  // batch
    int by = blockIdx.y;  // head
    int tx = threadIdx.x; // índice da linha
    int ty = threadIdx.y; // índice da coluna

    int qkv_offset = (bx * gridDim.y * seq_len * embed_dim) + (by * seq_len * embed_dim);
    int lm_offset = (bx * gridDim.y * seq_len) + (by * seq_len);

    extern __shared__ float s[];
    float* Q_tile = s;
    float* K_tile = Q_tile + block_size_row * embed_dim;
    float* V_tile = K_tile + block_size_col * embed_dim;
    float* S_tile = V_tile + block_size_col * embed_dim;

    const float eps = 1e-10f;
    int total_row_blocks = (seq_len + block_size_row - 1) / block_size_row;
    int total_col_blocks = (seq_len + block_size_col - 1) / block_size_col;

    for (int row_block = 0; row_block < total_row_blocks; row_block++) {
        int row_idx = row_block * block_size_row + tx;
        if (tx < block_size_row && row_idx < seq_len) {
            for (int d = 0; d < embed_dim; d++) {
                Q_tile[tx * embed_dim + d] = Q[qkv_offset + row_idx * embed_dim + d];
            }
        }

        float mi = (row_idx < seq_len) ? m[lm_offset + row_idx] : -INFINITY;
        float li = (row_idx < seq_len) ? l[lm_offset + row_idx] : 0.0f;

        for (int col_block = 0; col_block < total_col_blocks; col_block++) {
            int col_idx = col_block * blockDim.y + ty;
            if (ty < blockDim.y && col_idx < seq_len) {
                for (int d = 0; d < embed_dim; d++) {
                    K_tile[ty * embed_dim + d] = K[qkv_offset + col_idx * embed_dim + d];
                    V_tile[ty * embed_dim + d] = V[qkv_offset + col_idx * embed_dim + d];
                }
            }
            block.sync();

            if (tx < block_size_row && row_idx < seq_len) {
                float row_max = -INFINITY;
                float row_sum = 0.0f;
                // Calcular S = Q · K^T em blocos
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    float sum = 0.0f;
                    for (int d = 0; d < embed_dim; d++) {
                        sum += Q_tile[tx * embed_dim + d] * K_tile[j * embed_dim + d];
                    }
                    sum *= scale;
                    S_tile[tx * block_size_col + j] = sum;
                    row_max = fmaxf(row_max, sum);
                }

                // Softmax incremental
                float m_new = fmaxf(mi, row_max);
                float l_new = expf(mi - m_new) * li + expf(row_max - m_new);
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    S_tile[tx * block_size_col + j] = expf(S_tile[tx * block_size_col + j] - m_new);
                    row_sum += S_tile[tx * block_size_col + j];
                }
                l_new *= row_sum;

                // Atualizar O incrementalmente
                for (int d = 0; d < embed_dim; d++) {
                    float acc = 0.0f;
                    for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                        acc += S_tile[tx * block_size_col + j] * V_tile[j * embed_dim + d];
                    }
                    int out_idx = qkv_offset + row_idx * embed_dim + d;
                    float o_prev = O[out_idx];
                    O[out_idx] = (li * expf(mi - m_new) * o_prev + expf(row_max - m_new) * acc) / (l_new + eps);
                }

                mi = m_new;
                li = l_new;
            }
            block.sync();

            if (tx < block_size_row && row_idx < seq_len) {
                m[lm_offset + row_idx] = mi;
                l[lm_offset + row_idx] = li;
            }
        }
    }
}

// Backward FlashAttention-2
__global__ void flash_attention_2_backward(
    const float* __restrict__ Q,
    const float* __restrict__ K,
    const float* __restrict__ V,
    const float* __restrict__ dO,
    const float* __restrict__ l,
    const float* __restrict__ m,
    float* __restrict__ dQ,
    float* __restrict__ dK,
    float* __restrict__ dV,
    const int seq_len,
    const int embed_dim,
    const int block_size_row,
    const int block_size_col,
    const float scale)
{
    cg::thread_block block = cg::this_thread_block();
    int bx = blockIdx.x;  // batch
    int by = blockIdx.y;  // head
    int tx = threadIdx.x; // índice da linha
    int ty = threadIdx.y; // índice da coluna

    int qkv_offset = (bx * gridDim.y * seq_len * embed_dim) + (by * seq_len * embed_dim);
    //int lm_offset = (bx * gridDim.y * seq_len) + (by * seq_len);

    extern __shared__ float s[];
    float* Q_tile = s;
    float* K_tile = Q_tile + block_size_row * embed_dim;
    float* V_tile = K_tile + block_size_col * embed_dim;
    float* dO_tile = V_tile + block_size_col * embed_dim;
    float* S_tile = dO_tile + block_size_row * embed_dim;

    const float eps = 1e-10f;
    int total_row_blocks = (seq_len + block_size_row - 1) / block_size_row;
    int total_col_blocks = (seq_len + block_size_col - 1) / block_size_col;

    for (int row_block = 0; row_block < total_row_blocks; row_block++) {
        int row_idx = row_block * block_size_row + tx;
        if (tx < block_size_row && row_idx < seq_len) {
            for (int d = 0; d < embed_dim; d++) {
                Q_tile[tx * embed_dim + d] = Q[qkv_offset + row_idx * embed_dim + d];
                dO_tile[tx * embed_dim + d] = dO[qkv_offset + row_idx * embed_dim + d];
            }
        }

        //float mi = (row_idx < seq_len) ? m[lm_offset + row_idx] : -INFINITY;
        //float li = (row_idx < seq_len) ? l[lm_offset + row_idx] : 0.0f;

        for (int col_block = 0; col_block < total_col_blocks; col_block++) {
            int col_idx = col_block * block_size_col + ty;
            if (ty < block_size_col && col_idx < seq_len) {
                for (int d = 0; d < embed_dim; d++) {
                    K_tile[ty * embed_dim + d] = K[qkv_offset + col_idx * embed_dim + d];
                    V_tile[ty * embed_dim + d] = V[qkv_offset + col_idx * embed_dim + d];
                }
            }
            block.sync();

            if (tx < block_size_row && row_idx < seq_len) {
                float row_max = -INFINITY;
                // Recomputar S
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    float sum = 0.0f;
                    for (int d = 0; d < embed_dim; d++) {
                        sum += Q_tile[tx * embed_dim + d] * K_tile[j * embed_dim + d];
                    }
                    sum *= scale;
                    S_tile[tx * block_size_col + j] = sum;
                    row_max = fmaxf(row_max, sum);
                }

                float row_sum = 0.0f;
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    float val = expf(S_tile[tx * block_size_col + j] - row_max);
                    S_tile[tx * block_size_col + j] = val;
                    row_sum += val;
                }
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    S_tile[tx * block_size_col + j] /= (row_sum + eps);
                }

                // Calcular dS
                float sum_dS = 0.0f;
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    float ds = 0.0f;
                    for (int d = 0; d < embed_dim; d++) {
                        ds += dO_tile[tx * embed_dim + d] * V_tile[j * embed_dim + d];
                    }
                    sum_dS += ds * S_tile[tx * block_size_col + j];
                }
                for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                    float ds = 0.0f;
                    for (int d = 0; d < embed_dim; d++) {
                        ds += dO_tile[tx * embed_dim + d] * V_tile[j * embed_dim + d];
                    }
                    S_tile[tx * block_size_col + j] = scale * S_tile[tx * block_size_col + j] * (ds - sum_dS);
                }

                // Calcular dQ
                for (int d = 0; d < embed_dim; d++) {
                    float acc = 0.0f;
                    for (int j = 0; j < block_size_col && (col_block * block_size_col + j) < seq_len; j++) {
                        acc += S_tile[tx * block_size_col + j] * K_tile[j * embed_dim + d];
                    }
                    int idx = qkv_offset + row_idx * embed_dim + d;
                    atomicAdd(&dQ[idx], acc);
                }
            }
            block.sync();

            if (ty < block_size_col && col_idx < seq_len) {
                for (int d = 0; d < embed_dim; d++) {
                    float dk_acc = 0.0f;
                    float dv_acc = 0.0f;
                    for (int r = 0; r < block_size_row && (row_block * block_size_row + r) < seq_len; r++) {
                        float ds = S_tile[r * block_size_col + ty];
                        dk_acc += ds * Q_tile[r * embed_dim + d];
                        dv_acc += S_tile[r * block_size_col + ty] * dO_tile[r * embed_dim + d];
                    }
                    int idx = qkv_offset + col_idx * embed_dim + d;
                    atomicAdd(&dK[idx], dk_acc);
                    atomicAdd(&dV[idx], dv_acc);
                }
            }
            block.sync();
        }
    }
}

template <typename T>
struct DeviceArray {
    T* ptr;
    size_t size;

    DeviceArray(size_t s, bool zero_init = false) : size(s) {
        CUDA_CHECK(hipMalloc(&ptr, size));
        if (zero_init) CUDA_CHECK(hipMemset(ptr, 0, size));
    }

    ~DeviceArray() { hipFree(ptr); }

    void hiprand_init(float min_val = 0.01f, float max_val = 0.1f) {
        hiprandGenerator_t gen;
        CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, time(0)));
        CURAND_CHECK(hiprandGenerateUniform(gen, ptr, size / sizeof(T)));

        int n = size / sizeof(T);
        float* temp;
        CUDA_CHECK(hipMalloc(&temp, size));
        CUDA_CHECK(hipMemcpy(temp, ptr, size, hipMemcpyDeviceToDevice));

        int threads = 256;
        int blocks = (n + threads - 1) / threads;
        scale_shift_kernel<<<blocks, threads>>>(temp, ptr, n, min_val, max_val - min_val);
        CUDA_CHECK(hipGetLastError());

        CUDA_CHECK(hipFree(temp));
        CURAND_CHECK(hiprandDestroyGenerator(gen));
    }
};

void print_matrix(float* matrix, int batch_size, int num_heads, int seq_len, int embed_dim, const char* name) {
    float* host_matrix = new float[batch_size * num_heads * seq_len * embed_dim];
    CUDA_CHECK(hipMemcpy(host_matrix, matrix, batch_size * num_heads * seq_len * embed_dim * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "\n" << name << ":\n";
    for (int b = 0; b < batch_size; b++) {
        std::cout << "Batch " << b << ":\n";
        for (int h = 0; h < num_heads; h++) {
            std::cout << "Head " << h << ":\n";
            for (int i = 0; i < seq_len; i++) {
                for (int j = 0; j < embed_dim; j++) {
                    int idx = b * num_heads * seq_len * embed_dim + h * seq_len * embed_dim + i * embed_dim + j;
                    std::cout << host_matrix[idx] << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }
    }
    delete[] host_matrix;
}

float dotProduct(const float* a, const float* b, size_t length) {
    float produto = 0.0f;
    for (size_t i = 0; i < length; ++i)
        produto += a[i] * b[i];
    return produto;
}

float* softmax(const float* scores, size_t n) {
    float* exp_scores = new float[n];
    float soma_exp = 0.0f;
    for (size_t i = 0; i < n; ++i) {
        exp_scores[i] = std::exp(scores[i]);
        soma_exp += exp_scores[i];
    }
    for (size_t i = 0; i < n; ++i)
        exp_scores[i] /= soma_exp;
    return exp_scores;
}

void attention(const float* queries, size_t num_queries, size_t dim,
               const float* keys, size_t num_keys,
               const float* values, size_t value_dim,
               float* output) {
    for (size_t i = 0; i < num_queries; ++i) {
        const float* query = queries + i * dim;
        float escala = std::sqrt(static_cast<float>(dim));

        float* scores = new float[num_keys];
        for (size_t j = 0; j < num_keys; ++j) {
            const float* key = keys + j * dim;
            scores[j] = dotProduct(query, key, dim) / escala;
        }

        float* pesos = softmax(scores, num_keys);

        float* out_query = output + i * value_dim;
        for (size_t k = 0; k < value_dim; ++k)
            out_query[k] = 0.0f;

        for (size_t j = 0; j < num_keys; ++j) {
            const float* value = values + j * value_dim;
            for (size_t k = 0; k < value_dim; ++k)
                out_query[k] += pesos[j] * value[k];
        }

        delete[] scores;
        delete[] pesos;
    }
}

bool compareMatrices(const float* matA, const float* matB, size_t rows, size_t cols, float tolerance = 1e-5f) {
    size_t totalElements = rows * cols;
    for (size_t i = 0; i < totalElements; ++i) {
        if (std::fabs(matA[i] - matB[i]) > tolerance) {
            return false;
        }
    }
    return true;
}

int main() {
    constexpr int batch_size = 1;
    constexpr int num_heads = 1;
    constexpr int seq_len = 32;
    const int embed_dim = 32;
    float negative_infinity_host = -INFINITY;

    const float scale = 1.0f / sqrtf(embed_dim);

    size_t matrix_size = batch_size * num_heads * seq_len * embed_dim * sizeof(float);
    size_t vector_size = batch_size * num_heads * seq_len * sizeof(float);

    DeviceArray<float> Q(matrix_size); Q.hiprand_init(0.01f, 0.1f);
    DeviceArray<float> K(matrix_size); K.hiprand_init(0.01f, 0.1f);
    DeviceArray<float> V(matrix_size); V.hiprand_init(0.01f, 0.1f);
    DeviceArray<float> O(matrix_size, true);
    DeviceArray<float> l(vector_size, true);
    DeviceArray<float> m(vector_size);
    CUDA_CHECK(hipMemset(m.ptr, *reinterpret_cast<int*>(&negative_infinity_host), vector_size));

    DeviceArray<float> dO(matrix_size); dO.hiprand_init(0.01f, 0.1f);
    DeviceArray<float> dQ(matrix_size, true);
    DeviceArray<float> dK(matrix_size, true);
    DeviceArray<float> dV(matrix_size, true);

    dim3 blockDim(16,16);
    dim3 gridDim((embed_dim + blockDim.x - 1)/blockDim.x, (seq_len + blockDim.y - 1)/blockDim.y);
    dim3 block(blockDim.x, blockDim.y);
    size_t smem_size = (blockDim.x * embed_dim +
                        2 * blockDim.y * embed_dim +
                        blockDim.x * blockDim.y) * sizeof(float);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    //print_matrix(Q.ptr, batch_size, num_heads, seq_len, embed_dim, "Q");
    //print_matrix(K.ptr, batch_size, num_heads, seq_len, embed_dim, "K");
    //print_matrix(V.ptr, batch_size, num_heads, seq_len, embed_dim, "V");

    CUDA_CHECK(hipEventRecord(start));
    flash_attention_2_forward<<<gridDim, blockDim, smem_size>>>(
        Q.ptr, K.ptr, V.ptr, O.ptr, l.ptr, m.ptr,
        seq_len, embed_dim, blockDim.x, blockDim.y, scale
    );
    CUDA_CHECK(hipDeviceSynchronize());

    float output[seq_len * embed_dim];
    float* query_matrix = new float[batch_size * num_heads * seq_len * embed_dim];
    float* key_matrix = new float[batch_size * num_heads * seq_len * embed_dim];
    float* value_matrix = new float[batch_size * num_heads * seq_len * embed_dim];
    float* output_matrix = new float[batch_size * num_heads * seq_len * embed_dim];
    CUDA_CHECK(hipMemcpy(query_matrix, Q.ptr, batch_size * num_heads * seq_len * embed_dim * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(key_matrix, K.ptr, batch_size * num_heads * seq_len * embed_dim * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(value_matrix, V.ptr, batch_size * num_heads * seq_len * embed_dim * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(output_matrix, O.ptr, batch_size * num_heads * seq_len * embed_dim * sizeof(float), hipMemcpyDeviceToHost));

    attention(query_matrix, seq_len, embed_dim, key_matrix, seq_len, value_matrix, embed_dim, output);

    if (compareMatrices(output_matrix, output, seq_len, embed_dim)) {
        std::cout << "matrix1 e matrix2 são iguais." << std::endl;
    } else {
        std::cout << "matrix1 e matrix2 são diferentes." << std::endl;
    }


    print_matrix(O.ptr, batch_size, num_heads, seq_len, embed_dim, "Output (O)");

    for (size_t i = 0; i < seq_len; i++) {
        std::cout << "Resultado da atenção para a query " << i + 1 << ": ";
        for (size_t j = 0; j < embed_dim; j++) {
            std::cout << output[i * embed_dim + j] << " ";
        }
        std::cout << std::endl;
    }
    //print_matrix(dO.ptr, batch_size, num_heads, seq_len, embed_dim, "dO");

    /* flash_attention_2_backward<<<grid, block, smem_size>>>(
        Q.ptr, K.ptr, V.ptr, dO.ptr, l.ptr, m.ptr,
        dQ.ptr, dK.ptr, dV.ptr,
        seq_len, embed_dim, blockDim.x, blockDim.y, scale
    );
    CUDA_CHECK(hipDeviceSynchronize()); */

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float ms;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    std::cout << "Tempo total de execução (forward + backward): " << ms << " ms\n";

    //print_matrix(dQ.ptr, batch_size, num_heads, seq_len, embed_dim, "dQ");
    //print_matrix(dK.ptr, batch_size, num_heads, seq_len, embed_dim, "dK");
    //print_matrix(dV.ptr, batch_size, num_heads, seq_len, embed_dim, "dV");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    //delete[] query_matrix;
    //delete[] key_matrix;
    //delete[] value_matrix;
    //delete[] output_matrix;

    return 0;
}
