
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void Soma(int *a, int  *b, int *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<n) c[i] =  a[i] + b[i];
}


int main(){
    int n, x; 
    cin >> n;
    int *a = new int[n];
    int *b = new int[n];
    int *c = new int[n];

    for (int i = 0; i < n; i++) {
        cin >> x;
        a[i] = x;
    }
    for (int i = 0; i < n; i++) {
        cin >> x;
        b[i] = x;
    }

    int *pa, *pb, *pc;
    hipMalloc(&pa, n*sizeof(int));
    hipMalloc(&pb, n*sizeof(int));
    hipMalloc(&pc, n*sizeof(int));
    hipMemcpy(pa, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pb, b, n*sizeof(int), hipMemcpyHostToDevice);
    
    int blocksize = 256;
    int gridsize = (n + blocksize - 1) / blocksize; 
    Soma<<<gridsize, blocksize>>>(pa, pb, pc, n);
    hipDeviceSynchronize();

    hipMemcpy(c, pc, n*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        cout << c[i] << " ";
    }

    hipFree(pa);
    hipFree(pb);
    hipFree(pc);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
